#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <chrono>
#include<stdlib.h>
using namespace std;

__global__ void matrix(int *a, int*b, int*c, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i <=N){

  }
}
int main(int argc, char** argv) {
  int size, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  const int N = 256;
  const int M = 64;
  vector<float> A(N*N);
  vector<float> B(N*N);
  vector<float> C(N*N, 0);
  float subA[N*N/size];
  float subB[N*N/size];
  float subC[N*N/size];
  int gpusize, gpurank ;

  float *a;
  float *b;
  float *c;
  //initial original matrix
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      A[N*i+j] = drand48();
      B[N*i+j] = drand48();
    }
  }

//initialize sub matrix for MPI
  int offset = N/size*rank;
  for (int i=0; i<N/size; i++)
    for (int j=0; j<N; j++)
      subA[N*i+j] = A[N*(i+offset)+j];
  for (int i=0; i<N; i++)
    for (int j=0; j<N/size; j++)
      subB[N/size*i+j] = B[N*i+j+offset];
  int recv_from = (rank + 1) % size;
  int send_to = (rank - 1 + size) % size;

//start parallerithm
  hipGetDeviceCount(&gpusize);
  hipSetDevice(rank % gpusize);
  hipGetDevice(&gpurank);
  hipMalloc(&a, N*sizeof(float));
  hipMalloc(&b, N*sizeof(float));
  hipMalloc(&c, N*sizeof(float));
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);

hipMemcpy(a,subA,N*sizeof(float),hipMemcpyHostToDevice);
hipEventRecord(stop);
hipEventSynchronize(stop);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
  double comp_time = 0, comm_time = 0;
  for(int irank=0; irank<size; irank++) {
    MPI_Barrier(MPI_COMM_WORLD);
    auto tic = chrono::steady_clock::now();
    offset = N/size*((rank+irank) % size);
    for (int i=0; i<N/size; i++)
      for (int j=0; j<N/size; j++)
        for (int k=0; k<N; k++)
          subC[N*i+j+offset] += subA[N*i+k] * subB[N/size*k+j];
    auto toc = chrono::steady_clock::now();
    comp_time += chrono::duration<double>(toc - tic).count();
    //hipMemcpy(c,subC,N*N/size*sizeof(float),hipMemcpyDeviceToHost);

    MPI_Send(&subB[0], N*N/size, MPI_FLOAT, send_to, 0, MPI_COMM_WORLD);
    MPI_Recv(&subB[0], N*N/size, MPI_FLOAT, recv_from, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    tic = chrono::steady_clock::now();
    comm_time += chrono::duration<double>(tic - toc).count();
  }
  MPI_Allgather(&subC[0], N*N/size, MPI_FLOAT, &C[0], N*N/size, MPI_FLOAT, MPI_COMM_WORLD);
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      for (int k=0; k<N; k++)
        C[N*i+j] -= A[N*i+k] * B[N*k+j];
  double err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(C[N*i+j]);
  if(rank==0) {
    double time = comp_time+comm_time;
    printf("N    : %d\n",N);
    printf("comp : %lf s\n", comp_time);
    printf("comm : %lf s\n", comm_time);
    printf("total: %lf s (%lf GFlops)\n",time,2.*N*N*N/time/1e9);
    printf("error: %lf\n",err/N/N);
    printf("cuda time: %lf\n",milliseconds);
    printf("cuda est : %lf\n",milliseconds*(N/size)*(N/size));
  }
  hipFree(a);
  //hipFree(b);
  //hipFree(c);
  MPI_Finalize();

}
