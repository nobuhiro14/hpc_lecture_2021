#include <mpi.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <chrono>
using namespace std;

nt main(void) {
  const int N = 2000;
  const int M = 1024;
  float *a;
  vector<float> subA(N);
  for(int i = 0;i<N;i++)
    subA[i] = 1;
//hipMalloc((void**)&a,N*(float));
//hipMemcpy(a,subA,N*(float),hipMemcpyHostToDevice);


  float A(N);
  for(int i = 0;i<N;i++)
    A[i] = 1;

  hipMalloc((void**)&a,N*(float));
  hipMemcpy(a,A,N*(float),hipMemcpyHostToDevice);



  hipFree(a);
  
}
